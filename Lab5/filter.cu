#include "hip/hip_runtime.h"
// Lab 5, image filters with CUDA.

// Compile with a command-line similar to Lab 4:
// nvcc filter.cu -c -arch=sm_30 -o filter.o
// g++ filter.o milli.c readppm.c -lGL -lm -lcuda -lcudart -L/usr/local/cuda/lib -lglut -o filter
// or (multicore lab)
// nvcc filter.cu -c -arch=sm_20 -o filter.o
// g++ filter.o milli.c readppm.c -lGL -lm -lcuda -L/usr/local/cuda/lib64 -lcudart -lglut -o filter

// 2017-11-27: Early pre-release, dubbed "beta".
// 2017-12-03: First official version! Brand new lab 5 based on the old lab 6.
// Better variable names, better prepared for some lab tasks. More changes may come
// but I call this version 1.0b2.
// 2017-12-04: Two fixes: Added command-lines (above), fixed a bug in computeImages
// that allocated too much memory. b3
// 2017-12-04: More fixes: Tightened up the kernel with edge clamping.
// Less code, nicer result (no borders). Cleaned up some messed up X and Y. b4
// 2022-12-07: A correction for a deprecated function.

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#ifdef __APPLE__
  #include <GLUT/glut.h>
  #include <OpenGL/gl.h>
#else
  #include <GL/glut.h>
#endif
#include "readppm.h"
#include "milli.h"

// Use these for setting shared memory size.
#define maxKernelSizeX 10
#define maxKernelSizeY 10

__global__ void filter(unsigned char *image, unsigned char *out, const unsigned int imagesizex, const unsigned int imagesizey, const int kernelsizex, const int kernelsizey)
{
	__shared__ unsigned char sharedMem[maxKernelSizeX * maxKernelSizeY * 3];

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	int dy, dx;
	unsigned int sumx, sumy, sumz;

	int divby = (2 * kernelsizex + 1) * (2 * kernelsizey + 1);

	// Load pixels into shared memory
	int sharedIndex = threadIdx.y * blockDim.x * 3 + threadIdx.x * 3;
	int imageIndex = (y * imagesizex + x) * 3;
	for (int i = 0; i < 3; ++i)
	{
		sharedMem[sharedIndex + i] = image[imageIndex + i];
	}

	__syncthreads(); // Ensure all threads have loaded their data

	if (x < imagesizex && y < imagesizey)
	{
		sumx = 0;
		sumy = 0;
		sumz = 0;

		for (dy = -kernelsizey; dy <= kernelsizey; dy++)
		{
			for (dx = -kernelsizex; dx <= kernelsizex; dx++)
			{
				int sharedY = threadIdx.y + dy;
				int sharedX = threadIdx.x + dx;

				// Use max and min to avoid branching!
				int yy = min(max(y + dy, 0), imagesizey - 1);
				int xx = min(max(x + dx, 0), imagesizex - 1);

				int sharedIndex = sharedY * blockDim.x * 3 + sharedX * 3;
				sumx += sharedMem[sharedIndex + 0];
				sumy += sharedMem[sharedIndex + 1];
				sumz += sharedMem[sharedIndex + 2];
			}
		}

		out[imageIndex + 0] = sumx / divby;
		out[imageIndex + 1] = sumy / divby;
		out[imageIndex + 2] = sumz / divby;
	}
}

// Global variables for image data

unsigned char *image, *pixels, *dev_bitmap, *dev_input;
unsigned int imagesizey, imagesizex; // Image size

////////////////////////////////////////////////////////////////////////////////
// main computation function
////////////////////////////////////////////////////////////////////////////////
void computeImages(int kernelsizex, int kernelsizey)
{
	if (kernelsizex > maxKernelSizeX || kernelsizey > maxKernelSizeY)
	{
		printf("Kernel size out of bounds!\n");
		return;
	}

	pixels = (unsigned char *) malloc(imagesizex*imagesizey*3);
	hipMalloc( (void**)&dev_input, imagesizex*imagesizey*3);
	hipMemcpy( dev_input, image, imagesizey*imagesizex*3, hipMemcpyHostToDevice );
	hipMalloc( (void**)&dev_bitmap, imagesizex*imagesizey*3);
	dim3 blockDim(16, 16);
	dim3 grid((imagesizex + blockDim.x - 1) / blockDim.x, (imagesizey + blockDim.y - 1) / blockDim.y);
	filter<<<grid, blockDim>>>(dev_input, dev_bitmap, imagesizex, imagesizey, kernelsizex, kernelsizey);

	hipDeviceSynchronize();
//	Check for errors!
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
	hipMemcpy( pixels, dev_bitmap, imagesizey*imagesizex*3, hipMemcpyDeviceToHost );
	hipFree( dev_bitmap );
	hipFree( dev_input );
}

// Display images
void Draw()
{
// Dump the whole picture onto the screen.	
	glClearColor( 0.0, 0.0, 0.0, 1.0 );
	glClear( GL_COLOR_BUFFER_BIT );

	if (imagesizey >= imagesizex)
	{ // Not wide - probably square. Original left, result right.
		glRasterPos2f(-1, -1);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, image );
		glRasterPos2i(0, -1);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE,  pixels);
	}
	else
	{ // Wide image! Original on top, result below.
		glRasterPos2f(-1, -1);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, pixels );
		glRasterPos2i(-1, 0);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, image );
	}
	glFlush();
}

// Main program, inits
int main( int argc, char** argv) 
{
	glutInit(&argc, argv);
	glutInitDisplayMode( GLUT_SINGLE | GLUT_RGBA );

	if (argc > 1)
		image = readppm(argv[1], (int *)&imagesizex, (int *)&imagesizey);
	else
		image = readppm((char *)"maskros512.ppm", (int *)&imagesizex, (int *)&imagesizey);

	if (imagesizey >= imagesizex)
		glutInitWindowSize( imagesizex*2, imagesizey );
	else
		glutInitWindowSize( imagesizex, imagesizey*2 );
	glutCreateWindow("Lab 5");
	glutDisplayFunc(Draw);

	ResetMilli();

	computeImages(2, 2);

// You can save the result to a file like this:
//	writeppm("out.ppm", imagesizey, imagesizex, pixels);

	glutMainLoop();
	return 0;
}
