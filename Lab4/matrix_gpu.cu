#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void matrixAddition(float* A, float* B, float* C, int N) {
    int iy = blockIdx.x * blockDim.x + threadIdx.x;
    int ix = blockIdx.y * blockDim.y + threadIdx.y;
    int index = ix+iy*N;
    if (ix < N && iy < N)
    {
        C[index] = A[index] + B[index];
    }
}

void initializeMatrix(float* matrix, int N) {
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            matrix[i + N * j] = i + j;  
        }
    }
}

void printMatrix(float* matrix, int N) {
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            cout << matrix[i * N + j] << "  ";
        }
        cout << endl;
    }
}

int main() {
    const int N = 1024;  

    
    float* h_A = new float[N*N];
    float* h_B = new float[N*N];
    initializeMatrix(h_A, N);
    initializeMatrix(h_B, N);
    // printMatrix(h_A, N);
    cout << "---------------------------------------------------------" << endl;
    // printMatrix(h_B, N);
    cout << "---------------------------------------------------------" << endl;

    float *h_C = new float[N * N];

    float* d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N * N * sizeof(float));
    hipMalloc((void**)&d_B, N * N * sizeof(float));
    hipMalloc((void**)&d_C, N * N * sizeof(float));

    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(32, 32);
    dim3 gridSize((N+blockSize.x-1)/blockSize.x, (N+blockSize.y-1)/blockSize.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    matrixAddition<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
    {
        cout << "Error: " << hipGetErrorString(err) << endl;
    }
    hipEventRecord(stop);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << "Kernel Execution Time: " << milliseconds << " ms" << endl;

    hipMemcpy(h_C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // printMatrix(h_C, N);

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}