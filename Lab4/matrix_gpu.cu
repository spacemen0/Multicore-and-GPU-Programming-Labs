#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void matrixAddition(float* A, float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        int index = row * N + col;
        C[index] = A[index] + B[index];
    }
}

void initializeMatrix(float* matrix, int N) {
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            matrix[i * N + j] = i + j;  // You can set any initialization logic here
        }
    }
}

void printMatrix(float* matrix, int N) {
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            cout << matrix[i * N + j] << "  ";
        }
        cout << endl;
    }
}

int main() {
    const int N = 16;  // Matrix columns

    // Allocate and initialize matrices A and B on the host
    float* h_A = new float[N * N];
    float* h_B = new float[N * N];
    initializeMatrix(h_A, N);
    initializeMatrix(h_B, N);
    printMatrix(h_A, N);
    cout << "---------------------------------------------------------" << endl;
    printMatrix(h_B, N);
    cout << "---------------------------------------------------------" << endl;

    // Allocate matrix C to store the result on the host
    float* h_C = new float[N * N];

    // Allocate device (GPU) memory
    float* d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N * N * sizeof(float));
    hipMalloc((void**)&d_B, N * N * sizeof(float));
    hipMalloc((void**)&d_C, N * N * sizeof(float));

    // Copy input matrices from host to device
    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockSize(N, N);
    dim3 gridSize(1, 1);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start);

    // Launch the kernel on the GPU
    matrixAddition<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);

    // Record the stop event
    hipEventRecord(stop);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);

    // Calculate the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << "Kernel Execution Time: " << milliseconds << " ms" << endl;

    // Copy the result back from the GPU to the CPU
    hipMemcpy(h_C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // ... Handle the result stored in h_C ...
    printMatrix(h_C, N);

    // Free host memory
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}