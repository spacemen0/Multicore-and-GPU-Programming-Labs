// Simple CUDA example by Ingemar Ragnemalm 2009. Simplest possible?
// Assigns every element in an array with its index.
// Update 2022: Changed to cudaDeviceSynchronize.

// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

const int N = 100;
const int blocksize = 100;

__global__ void simple(float *c)
{
	c[threadIdx.x] = threadIdx.x;
}
__global__ void ssqrt(float *i)
{
	i[threadIdx.x] = sqrt((float)threadIdx.x);
}

int main()
{

	const int size = N * sizeof(float);
	dim3 dimBlock(blocksize, 1);
	dim3 dimGrid(1, 1);


	float *s = new float[N];
	float *sd;
	hipMalloc((void **)&sd, size);

	ssqrt<<<dimGrid, dimBlock>>>(sd);

	hipMemcpy(s, sd, size, hipMemcpyDeviceToHost);

	hipFree(sd);


	for (int i = 0; i < N; i++)
		printf("%f ", s[i]);
	printf("\n");
	delete[] s;
	printf("done\n");
	return EXIT_SUCCESS;
}
